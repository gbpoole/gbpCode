#include "hip/hip_runtime.h"
#include <gbpCommon.h>
#include <gbpSID.h>

__global__ void calc_array_multiply_kernel_int(int *a,int *b){
   a[threadIdx.x]*=b[threadIdx.x];
}

__global__ void calc_array_multiply_kernel_size_t(size_t *a,size_t *b){
   a[threadIdx.x]*=b[threadIdx.x];
}
__global__ void calc_array_multiply_kernel_float(float *a,float *b){
   a[threadIdx.x]*=b[threadIdx.x];
}
/*
__global__ void calc_array_multiply_kernel_double(double *a,double *b){
   a[threadIdx.x]*=b[threadIdx.x];
}
*/
void calc_array_multiply(void         *data_1,
                         void         *data_2,
                         void         *result,
    	                 size_t        n_data,
                         SID_Datatype  type,
                         int           mode){
  int datatype_size;

  // Don't do anything if there's no data
  if(n_data<1){
    if(type==SID_DOUBLE || check_mode_for_flag(mode,CALC_MODE_RETURN_DOUBLE))
      ((double *)result)[0]=0.;
    else if(type==SID_FLOAT)
      ((float  *)result)[0]=0.;
    else if(type==SID_INT)
      ((int    *)result)[0]=0;
    else if(type==SID_SIZE_T)
      ((size_t *)result)[0]=0;
    else
      SID_trap_error("Unknown variable type in calc_array_multiply",ERROR_LOGIC);
    return;
  }

  // Set size of datatype
  SID_Type_size(type,&datatype_size);

  // Define grid and block size
  int numBlocks          = 1;
  int numThreadsPerBlock = n_data;

  // Create data buffer on Device
  void   *data_1_device;
  void   *data_2_device;
  size_t  memSize = numBlocks*numThreadsPerBlock*(size_t)datatype_size;
  hipMalloc(SID_FARG data_1_device,memSize);
  hipMalloc(SID_FARG data_2_device,memSize);

  // Load data onto device
  hipMemcpy(data_1_device,data_1,memSize,hipMemcpyHostToDevice);
  hipMemcpy(data_2_device,data_2,memSize,hipMemcpyHostToDevice);

  //dim3 dimGrid(  );
  //dim3 dimBlock(  );

  if(type==SID_FLOAT)
     calc_array_multiply_kernel_float<<<numBlocks,numThreadsPerBlock>>>((float *)data_1_device,(float *)data_2_device);
/*
  else if(type==SID_DOUBLE)
     calc_array_multiply_kernel_double<<<numBlocks,numThreadsPerBlock>>>((double *)data_1_device,(double *)data_2_device);
*/
  else if(type==SID_INT)
     calc_array_multiply_kernel_int<<<numBlocks,numThreadsPerBlock>>>((int *)data_1_device,(int *)data_2_device);
  else if(type==SID_SIZE_T)
     calc_array_multiply_kernel_size_t<<<numBlocks,numThreadsPerBlock>>>((size_t *)data_1_device,(size_t *)data_2_device);
  else
    SID_trap_error("Unknown variable type in calc_array_multiply",ERROR_LOGIC);

  // Block until the Device is done
  hipDeviceSynchronize();

  // Copy the result back
  if(result==NULL)
    hipMemcpy(data_1,data_1_device,memSize,hipMemcpyDeviceToHost);
  else
    hipMemcpy(result,data_1_device,memSize,hipMemcpyDeviceToHost);

  // Free device buffer
  hipFree(data_1_device);
  hipFree(data_2_device);

  return;
}

