#include "hip/hip_runtime.h"
#include <gbpCommon.h>
#include <gbpSID.h>

__global__ void calc_array_multiply_kernel_int(int *a,int *b){
   int index=blockIdx.x*blockDim.x+threadIdx.x;
   a[index]*=b[index];
}
__global__ void calc_array_multiply_kernel_float(float *a,float *b){
   int index=blockIdx.x*blockDim.x+threadIdx.x;
   a[index]*=b[index];
}

void calc_array_multiply(void         *data_1,
                         void         *data_2,
                         void         *result,
    	                 size_t        n_data,
                         SID_Datatype  type,
                         int           mode){

  // Don't do anything if there's no data
  if(n_data<1)
    return;

  // Set size of datatype
  int datatype_size;
  SID_Type_size(type,&datatype_size);

  // Define grid and block size
  int numThreadsPerBlock = SID_CUDA_IDEAL_THREADS_PER_BLOCK;
  int numBlocks          = n_data/numThreadsPerBlock;
  if((n_data%numThreadsPerBlock)!=0) numBlocks++;
  //dim3 dimGrid(  );
  //dim3 dimBlock(  );

  // Create data buffer on Device
  void   *data_1_device;
  void   *data_2_device;
  size_t  memSize = n_data*(size_t)datatype_size;
  hipMalloc(SID_FARG data_1_device,memSize);
  hipMalloc(SID_FARG data_2_device,memSize);

  // Load data onto device
  hipMemcpy(data_1_device,data_1,memSize,hipMemcpyHostToDevice);
  hipMemcpy(data_2_device,data_2,memSize,hipMemcpyHostToDevice);

  if(type==SID_FLOAT)
     calc_array_multiply_kernel_float<<<numBlocks,numThreadsPerBlock>>>((float *)data_1_device,(float *)data_2_device);
  else if(type==SID_INT)
     calc_array_multiply_kernel_int<<<numBlocks,numThreadsPerBlock>>>((int *)data_1_device,(int *)data_2_device);
  else
    SID_trap_error("Unknown variable type in calc_array_multiply",ERROR_LOGIC);

  // Block until the Device is done
  hipDeviceSynchronize();

  hipError_t err = hipGetLastError();
  if(hipSuccess!=err)
     SID_trap_error("Cuda error: %s.", ERROR_LOGIC,hipGetErrorString( err) );

  // Copy the result back
  if(result==NULL)
    hipMemcpy(data_1,data_1_device,memSize,hipMemcpyDeviceToHost);
  else
    hipMemcpy(result,data_1_device,memSize,hipMemcpyDeviceToHost);

  // Free device buffer
  hipFree(data_1_device);
  hipFree(data_2_device);

  return;
}

